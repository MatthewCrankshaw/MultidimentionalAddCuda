
#include<iostream>
#include<vector>
#include<cmath>
#include<ctime>
#include<string>

using namespace std;

#include<hip/hip_runtime.h>

const long MAXDIM = 10;
const double RMIN = 2.0;
const double RMAX = 8.0;


//Global kernel code that runs on the device
__global__ void count_in(long *dev_index, bool *dev_isInSphereList, long *dev_count, long *dev_ndim, long *dev_halfb, double *dev_rsquare, long *dev_base){
	//dev_index is array of longs with length ndim
	//isInSphereList is array of bool with length ntotal

	// double rtestsq = 0;
	// bool inSphere = false; 

	// for(long k = 0; k < dev_ndim; ++k){
	// 	double xk = idx[k] - (*dev_halfb);
	// 	rtestsq += xk * xk;
	// }

	// if(rtestsq < (*dev_rsquare)) inSphere = true; 
	
	// //may produce error
	// //addone(dev_index, ndim, base, 0);

	// long newv = index[i] + 1;
	// if (newv >= base) {
	// 	index[i] = 0;
	// if (i < ndim - 1) 
	// 	addone(index, ndim, base, i+1);
	// }
	// else {
	// 	index[i] = newv;
	// }

	// long i = 1;
	// if(inSphere){
	// 	atomicAdd(dev_count, i);
	// }
}

void err_check(hipError_t err, char* text){
	if(err != hipSuccess){
		std::cout << "============================Cuda Error: " << hipGetErrorString(hipGetLastError()) << " in " << text << std::endl;
	}
}


//calculate the power using long integers
long powlong(long n, long k)
/* Evaluate n**k where both are long integers */
{
  long p = 1;
  for (long i = 0; i < k; ++i) p *= n;
  return p;
}

void count_in_seq(long *index, bool *sphereList, long pos,long ndim, long halfb, double rsquare, long base){ 
	
	//convert(pos, base, idx, ndim);

	long num = pos;

	for (long i = 0; i < ndim; ++i) index[i] = 0;
  	long idx = 0;
  	while (num != 0) {
    	long rem = num % base;
    	num = num / base;
    	index[idx] = rem;
    	++idx;
  	}

	double rtestsq = 0;
	bool inSphere = false; 

	for(long k = 0; k < ndim; ++k){
		double xk = index[k] - halfb;
		rtestsq += xk * xk;
	}

	if(rtestsq< rsquare) sphereList[pos] = true;
}


//========================================
// Main Function
//========================================

int main(int argc, char **argv){ 
  // You can make this larger if you want
  const long ntrials = 3;

  for (long n = 0; n < ntrials; ++n) {

    // Get a random value for the hypersphere radius between the two limits
    const double radius = drand48() * (RMAX - RMIN) + RMIN;

    // Get a random value for the number of dimensions between 1 and
    // MAXDIM inclusive
    const long  ndim = lrand48() % (MAXDIM - 1) + 1;
    std::cout << "Trial Number " << n << " Radius " << radius << " Dimensions " << ndim << " ... " << std::endl;

    long count = 0;

    const long halfb = static_cast<long>(floor(radius));
  	const long base = 2 * halfb + 1;
  	const double rsquare = radius * radius;
  	const long ntotal = powlong(base, ndim);

  	long *index = new long[ndim];
  	bool *isInSphereList = new bool[ntotal];

  	for(int i = 0; i < ndim; i++){
  		index[i] = 0;
  	}
  	for(int i = 0; i < ntotal; i++){
  		isInSphereList[i] = false;
  	}

    //CUDA part
    //=======================================================
  	//we need to split the problem into each pixel being an integer point

  	long *dev_index; 
  	bool *dev_isInSphereList;
  	long *dev_count;
  	long *dev_ndim; 
  	long *dev_halfb; 
  	double *dev_rsquare;
  	long *dev_base;

  	hipError_t err = hipMalloc((void**)&dev_index, sizeof(long)*ndim);
  	err_check(err, "index malloc");
  	err = hipMalloc((void**)&dev_isInSphereList, sizeof(bool)*ntotal);
  	err_check(err, "isInSphereList malloc");
  	err = hipMalloc((void**)&dev_count, sizeof(long));
  	err_check(err, "count malloc");
  	err = hipMalloc((void**)&dev_ndim, sizeof(long));
  	err_check(err, "ndim malloc");
  	err = hipMalloc((void**)&dev_halfb, sizeof(long));
  	err_check(err, "halfb malloc");
  	err = hipMalloc((void**)&dev_rsquare, sizeof(double));
  	err_check(err, "rsquare malloc");
  	err = hipMalloc((void**)&dev_base, sizeof(long));
  	err_check(err, "base malloc");

  	err = hipMemcpy(dev_index, index, sizeof(long)*ndim, hipMemcpyHostToDevice);
  	err_check(err, "index cpy");
  	err = hipMemcpy(dev_isInSphereList, isInSphereList, sizeof(bool)*ntotal, hipMemcpyHostToDevice);
  	err_check(err, "isinspherelist cpy");
  	err = hipMemcpy(dev_count, &count, sizeof(long), hipMemcpyHostToDevice);
  	err_check(err, "count cpy");
  	err = hipMemcpy(dev_ndim, &ndim, sizeof(long), hipMemcpyHostToDevice);
  	err_check(err, "ndim cpy");
  	err = hipMemcpy(dev_halfb, &halfb, sizeof(long), hipMemcpyHostToDevice);
  	err_check(err, "halfb cpy");
  	err = hipMemcpy(dev_rsquare, &rsquare, sizeof(double), hipMemcpyHostToDevice);
  	err_check(err, "rsquare cpy");
  	err = hipMemcpy(dev_base, &base, sizeof(long),hipMemcpyHostToDevice);
  	err_check(err, "base cpy");

  	int threadsPerBlock = 1024; 
  	int numBlocks = (ntotal + threadsPerBlock - 1) / threadsPerBlock;

  	cout << "Number of Threads Per Block: " << threadsPerBlock << endl;
  	cout << "Number of Blocks Per Grid: " << numBlocks << endl;
  	cout << "N Total: " << ntotal << endl;

  	count_in<<<threadsPerBlock, numBlocks>>>(dev_index, dev_isInSphereList, dev_count, dev_ndim, dev_halfb, dev_rsquare, dev_base);

  	err = hipMemcpy(&count, dev_count, sizeof(long), hipMemcpyDeviceToHost);
  	err_check(err, "count cpy to host");


  	//sequential
  	for(long n = 0; n < ntotal; ++n){
  		count_in_seq(index, isInSphereList, n, ndim , halfb, rsquare, base);
  	}

  	for(int i = 0; i < ntotal; i++){
  		if(isInSphereList[i]){
  			count ++;
  		}
  	}

    std::cout << " -> " << "count" << " " << count << "\n" << std::endl;

    free(index);
  }
}