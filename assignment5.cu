
#include<iostream>
#include<vector>
#include<cmath>
#include<ctime>
#include<string>

using namespace std;

#include<hip/hip_runtime.h>

#define ULL unsigned long long

const long MAXDIM = 10;
const double RMIN = 2.0;
const double RMAX = 8.0;




//Global kernel code that runs on the device
__global__ void count_in(ULL *dev_count, long dev_ntotal,long dev_ndim, long dev_halfb, double dev_rsquare, long dev_base){
	
	//Calculate the position of this kernel in the data
	ULL blockID = (blockIdx.y * gridDim.x) + blockIdx.x;
	ULL pos = (blockID * blockDim.x) + threadIdx.x;

	//If this threads position in the data is further than we need to calculate
	//Then we return
	if(pos >= dev_ntotal) return;

	double rtestsq = 0;
	long idx = 0;
	long index[MAXDIM];
	for (long i = 0; i < dev_ndim; ++i) index[i] = 0;
	
  	//Convert the decimal number into another base system
   	while (pos != 0) {
    	long rem = pos % dev_base;
     	pos = pos / dev_base;
     	index[idx] = rem;
    	++idx;
  	}


	for(long k = 0; k < dev_ndim; ++k){
	 	double xk = index[k] - dev_halfb;
	 	rtestsq += xk * xk;
	}

	//If the value is inside the sphere
	//Atomically add 1 to the count
	if(rtestsq < dev_rsquare){
		atomicAdd(dev_count, 1);
	}
}

void err_check(hipError_t err, char* text){
	if(err != hipSuccess){
		std::cout << "============================Cuda Error: " << hipGetErrorString(hipGetLastError()) << " in " << text << std::endl;
	}
}


//calculate the power using long integers
long powlong(long n, long k)
/* Evaluate n**k where both are long integers */
{
  long p = 1;
  for (long i = 0; i < k; ++i) p *= n;
  return p;
}
					
void count_in_seq(ULL &count,int thread, int block,long ndim, long halfb, double rsquare, long base){ 

	int num = (block * 1024) + thread;
	double rtestsq = 0;
	long idx = 0;

	long index[MAXDIM];

	for (long i = 0; i < ndim; ++i){
		index[i] = 0;
	}
  	
  	while (num != 0) {
    	long rem = num % base;
    	num = num / base;
    	index[idx] = rem;
    	++idx;
  	}

	for(long k = 0; k < ndim; ++k){
		double xk = index[k] - halfb;
		rtestsq += xk * xk;
	}

	if(rtestsq< rsquare){

		//This needs to be an atomic add
		count++;
	}
}


//========================================
// Main Function
//========================================

int main(int argc, char **argv){ 
  // You can make this larger if you want
  const long ntrials = 10;

  for (long n = 0; n < ntrials; ++n) {

    // Get a random value for the hypersphere radius between the two limits
    const double radius = drand48() * (RMAX - RMIN) + RMIN;

    // Get a random value for the number of dimensions between 1 and
    // MAXDIM inclusive
    const long  ndim = lrand48() % (MAXDIM - 1) + 1;
    std::cout << "Trial Number " << n << " Radius " << radius << " Dimensions " << ndim << " ... " << std::endl;


    //Count for counting the number of integer points that land in the circle
    ULL count = 0;

    //Set up the variable that will be needed by the cuda kernel
    const long halfb = static_cast<long>(floor(radius));
  	const long base = 2 * halfb + 1;
  	const double rsquare = radius * radius;
  	const long ntotal = powlong(base, ndim);


    //CUDA part
    //=======================================================
  	//we need to split the problem each integer point in the n dimentional space

  	// Set up the device count variable which we will need to retrieve later
  	ULL *dev_count;
  	hipMalloc((void**)&dev_count, sizeof(ULL));
  	hipMemcpy(dev_count, &count, sizeof(ULL), hipMemcpyHostToDevice);

 	//Set up the number of threads per block and blocks per grid
  	dim3 threadsPerBlock(1024, 1, 1);
  	dim3 blocksPerGrid(65535,7,1);


  	cout << "Number of Threads Per Block: " << threadsPerBlock.x 	<< " " << threadsPerBlock.y << " " << threadsPerBlock.z << endl;
  	cout << "Number of Blocks Per Grid: " 	<< blocksPerGrid.x 		<< " " << blocksPerGrid.y 	<< " " << blocksPerGrid.z 	<< endl;
  	cout << "N Total: " 					<< ntotal 				<< endl;

  	//Run the device kernel
  	count_in<<<blocksPerGrid, threadsPerBlock>>>(dev_count, ntotal, ndim, halfb, rsquare, base);


  	//Retrieve the memory from the device for the count
  	hipMemcpy(&count, dev_count, sizeof(unsigned long long), hipMemcpyDeviceToHost);

    std::cout << " -> " << "count" << " " << count << "\n" << std::endl;
  }
}